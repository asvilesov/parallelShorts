
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#define n 1024

__global__ void naive_mat_mul(int *a, int *b, int *c){
	int my_x = blockIdx.x*blockDim.x + threadIdx.x;
	int my_y = blockIdx.y*blockDim.y + threadIdx.y;

	int i = 0;
	int loc_c = 0;

	for(i = 0; i < n; i++){
		loc_c += a[my_x*n + i]*b[i*n + my_y];
	}

	c[my_x*n + my_y] = loc_c;  
}

int main(){	
	int i;
	hipEvent_t start, stop;
	float time_execute = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// matrix size is n*n
	//int b = 128; //block size
	//int b = atoi(argv[1]);

	
	int *A = (int*) malloc (sizeof(int)*n*n);
	int *B = (int*) malloc (sizeof(int)*n*n);
	int *C = (int*) malloc (sizeof(int)*n*n);
	for (i=0; i<n*n; i++) {
		A[i] = 1;
		B[i] = 2;
		C[i] = 0;
	}

	int *gpu_a, *gpu_b, *gpu_c;
	hipMalloc((void**)&gpu_a, sizeof(int)*n*n); 
	hipMalloc((void**)&gpu_b, sizeof(int)*n*n);
	hipMalloc((void**)&gpu_c, sizeof(int)*n*n);

	hipMemcpy(gpu_a, A, sizeof(int)*n*n, hipMemcpyHostToDevice);
	hipMemcpy(gpu_b, B, sizeof(int)*n*n, hipMemcpyHostToDevice);
	
	dim3 dimGrid(64,64);
	dim3 dimBlock(16,16);

	hipEventRecord(start, 0);

	naive_mat_mul<<< dimGrid, dimBlock >>> (gpu_a, gpu_b, gpu_c);

	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_execute, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(C, gpu_c, sizeof(int)*n*n, hipMemcpyDeviceToHost);

	printf("C[451][451] = %i\n", C[451*1024+451]);
	//printf("Total Time: %f\n", time_execute);

	free(A);
	free(B);
	free(C);
	hipFree(gpu_a);
	hipFree(gpu_b);
	hipFree(gpu_c);
	return 0;

}	