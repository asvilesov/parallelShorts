
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#define n 1024

__global__ void mat_mul(int *a, int *b, int *c){
	int my_x = blockIdx.x*blockDim.x + threadIdx.x;
	int my_y = blockIdx.y*blockDim.y + threadIdx.y;
	int i = 0;
	int j = 0;
	__shared__ int s_A[32][32];
	__shared__ int s_B[32][32];
	int loc_c = 0;


	for(i = 0; i < n/blockDim.x; i++){ //can be either block dim since they are the same
		s_A[threadIdx.x][threadIdx.y] = a[1024*my_x + i*blockDim.y + threadIdx.y];
		s_B[threadIdx.x][threadIdx.y] = b[1024*(threadIdx.x + i*blockDim.x) + my_y];

		__syncthreads();
		
		for(j = 0; j < blockDim.x; j++){
			loc_c += s_A[threadIdx.x][j]*s_B[j][threadIdx.y];
		}

		__syncthreads();
		
	}

	c[my_x*n + my_y] = loc_c;  
}

int main(){	
	int i;

	hipEvent_t start, stop;
	float time_execute = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	int *A = (int*) malloc (sizeof(int)*n*n);
	int *B = (int*) malloc (sizeof(int)*n*n);
	int *C = (int*) malloc (sizeof(int)*n*n);
	for (i=0; i<n*n; i++) {
		A[i] = 1;
		B[i] = 2;
		C[i] = 0;
	}

	int *gpu_a, *gpu_b, *gpu_c;
	hipMalloc((void**)&gpu_a, sizeof(int)*n*n); 
	hipMalloc((void**)&gpu_b, sizeof(int)*n*n);
	hipMalloc((void**)&gpu_c, sizeof(int)*n*n);

	hipMemcpy(gpu_a, A, sizeof(int)*n*n, hipMemcpyHostToDevice);
	hipMemcpy(gpu_b, B, sizeof(int)*n*n, hipMemcpyHostToDevice);
	
	dim3 dimGrid(32,32);
	dim3 dimBlock(32,32);

	hipEventRecord(start, 0);

	mat_mul<<< dimGrid, dimBlock >>> (gpu_a, gpu_b, gpu_c);

	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_execute, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(C, gpu_c, sizeof(int)*n*n, hipMemcpyDeviceToHost);

	printf("C[451][451] = %i\n", C[451*1024+451]);
	//printf("Total Time: %f\n", time_execute);
    

	free(A);
	free(B);
	free(C);
	hipFree(gpu_a);
	hipFree(gpu_b);
	hipFree(gpu_c);
	return 0;
	
}	